#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row,
                                         int32_t col) {
  return row * total_col + col;
}

__global__ void check_kernel(
    const int32_t* input,           // [num_vocab, max_sequence_len]
    const int32_t* sequence_len,    // [num_vocab]
    const int32_t* shift_table,         // [node_num, symbol_num, max_edge_num_for_same_t]
    const int32_t* edge_num_table,      // [node_num, symbol_num]
    const int32_t* push_table,          // [edge_num, max_push_len]
    const int32_t* pop_table,           // [edge_num, max_pop_len]
    const int32_t* dest_table,          // [edge_num]
    const int32_t* cur_stack,
    int32_t* state_list,
    int cur_stack_top,
    int num_seqs,
    int num_symbol,
    int num_node, 
    int max_seq_len,
    int max_edge_num_for_same_t,
    int max_stack_depth,
    int max_pop_len,
    int max_push_len
) {
    int seq_idx = blockIdx.x * blockDim.x + threadIdx.x;  // 每个线程对应一个输入序列
    int tid = threadIdx.x;
    if (seq_idx >= num_seqs) return;
    int seq_len = sequence_len[seq_idx];  // 输入序列长度

    int stack_top = 0;
    // extern __shared__ int32_t stack[];
    int32_t stack[64];

    int cur_state = state_list[seq_idx];

    // 初始化栈
    for (int i = 0; i < cur_stack_top; ++i) {
        stack[i] = cur_stack[i];  // 初始状态
    }
    stack_top = cur_stack_top;
    for (int i = 0; i < seq_len; ++i) {
        if (cur_state == -1) break;  // 如果状态已经变为无效，结束循环

        int input_symbol = input[seq_idx * max_seq_len + i];
        int edge_num = edge_num_table[index(num_symbol, cur_state, input_symbol)];
        int edge_start_index = cur_state * num_symbol * max_edge_num_for_same_t + input_symbol * max_edge_num_for_same_t;
        bool match = false;

        for (int edge_idx = 0; edge_idx < edge_num; ++edge_idx) {
            int edge = shift_table[edge_start_index + edge_idx];
            const int32_t* push_list = push_table + edge * max_push_len;
            const int32_t* pop_list = pop_table + edge * max_pop_len;
            int dest_state = dest_table[edge];

            // 检查栈顶是否匹配 pop_list
            bool check = true;
            for (int j = 0; j < max_pop_len && pop_list[j] != -1; ++j) {
                //if (stack_top - j - 1 < 0 || stack[index(max_stack_depth, tid, stack_top - j - 1)] != pop_list[j]) {
                if (stack_top - j - 1 < 0 || stack[stack_top - j - 1] != pop_list[j]) {
                    check = false;
                    break;
                }
            }
            if (check && !match) {
                cur_state = dest_state;
                // 执行 pop 操作
                for (int j = 0; j < max_pop_len && pop_list[j] != -1; ++j) {
                    --stack_top;
                }
                // 执行 push 操作
                for (int j = 0; j < max_push_len && push_list[j] != -1; ++j) {
                    // stack[index(max_stack_depth, tid, stack_top)] = push_list[j];
                    stack[stack_top] = push_list[j];
                    stack_top++;
                }

                match = true;
                break;
            }
        }

        if (!match) {
            cur_state = -1;  // 无法匹配，设置为无效状态
            break;
        }
    }

    // 存储最终的状态和栈指针
    state_list[seq_idx] = cur_state;
}

void check_dpda(
    torch::Tensor input_sequences,
    torch::Tensor sequence_len,
    torch::Tensor shift_table,
    torch::Tensor edge_num_table,
    torch::Tensor push_table,
    torch::Tensor pop_table,
    torch::Tensor dest_table,
    torch::Tensor current_stack,
    torch::Tensor start_state,
    int max_stack_depth
) {
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    const int num_seqs = input_sequences.size(0);
    const int max_seq_len = input_sequences.size(1);
    const int num_node = edge_num_table.size(0);
    const int num_symbol = edge_num_table.size(1);
    const int max_edge_num_for_same_t = shift_table.size(2);
    const int max_pop_len = pop_table.size(1);
    const int max_push_len = push_table.size(1);

    check_kernel<<<(num_seqs + 255) / 256, 256, 0, stream>>>(
        input_sequences.data_ptr<int>(),
        sequence_len.data_ptr<int>(),
        shift_table.data_ptr<int>(),
        edge_num_table.data_ptr<int>(),
        push_table.data_ptr<int>(),
        pop_table.data_ptr<int>(),
        dest_table.data_ptr<int>(),
        current_stack.data_ptr<int>(),
        start_state.data_ptr<int>(),
        current_stack.size(0),
        num_seqs,
        num_symbol,
        num_node,
        max_seq_len,
        max_edge_num_for_same_t,
        max_stack_depth,
        max_pop_len,
        max_push_len
    );
}

__global__ void batched_check_kernel(
    const int32_t* input,           // [num_seqs, max_sequence_len]
    const int32_t* sequence_len,    // [num_seqs]
    const int32_t* shift_table,         // [node_num, symbol_num, max_edge_num_for_same_t]
    const int32_t* edge_num_table,      // [node_num, symbol_num]
    const int32_t* push_table,          // [edge_num, max_push_len]
    const int32_t* pop_table,           // [edge_num, max_pop_len]
    const int32_t* dest_table,          // [edge_num]
    int32_t* cur_stack,
    int32_t* start_state_list,
    int32_t* cur_stack_top,
    int32_t* output,
    int num_batch,
    int num_seqs,
    int num_symbol,
    int num_node, 
    int max_seq_len,
    int max_edge_num_for_same_t,
    int max_stack_depth,
    int max_pop_len,
    int max_push_len,
    const int seq_per_thread,
    const int block_size
) {
    int batch_idx = blockIdx.x;
    int seq_idx_start = (blockIdx.y * block_size) + threadIdx.x * seq_per_thread;  // 每个线程对应一个输入序列
    int seq_idx_end = seq_idx_start + seq_per_thread;
    if (batch_idx >= num_batch) return;
    int32_t stack[64];
    // if(batch_idx == 0) printf("%d %d %d %d %d %d\n", batch_idx, seq_idx_start, seq_idx_end, blockIdx.y, threadIdx.x, blockDim.x);
    // extern __shared__ int32_t stack[];

    for (int seq_idx = seq_idx_start; seq_idx < seq_idx_end; seq_idx++) {
        if (seq_idx >= num_seqs) return;
        int seq_len = sequence_len[seq_idx];  // 输入序列长度
        int stack_top = cur_stack_top[batch_idx];
        
        int cur_state = start_state_list[batch_idx];
        // 初始化栈
        for (int i = 0; i < stack_top; ++i) {
            stack[i] = cur_stack[batch_idx * max_stack_depth + i];  // 初始状态
        }

        for (int i = 0; i < seq_len; ++i) {
            if (cur_state == -1) break;  // 如果状态已经变为无效，结束循环

            int input_symbol = input[seq_idx * max_seq_len + i];
            int edge_num = edge_num_table[index(num_symbol, cur_state, input_symbol)];
            int edge_start_index = cur_state * num_symbol * max_edge_num_for_same_t + input_symbol * max_edge_num_for_same_t;
            bool match = false;

            for (int edge_idx = 0; edge_idx < edge_num; ++edge_idx) {
                int edge = shift_table[edge_start_index + edge_idx];
                const int32_t* push_list = push_table + edge * max_push_len;
                const int32_t* pop_list = pop_table + edge * max_pop_len;
                int dest_state = dest_table[edge];

                // 检查栈顶是否匹配 pop_list
                bool check = true;
                for (int j = 0; j < max_pop_len && pop_list[j] != -1; ++j) {
                    //if (stack_top - j - 1 < 0 || stack[index(max_stack_depth, tid, stack_top - j - 1)] != pop_list[j]) {
                    if (stack_top - j - 1 < 0 || stack[stack_top - j - 1] != pop_list[j]) {
                        check = false;
                        break;
                    }
                }
                if (check && !match) {
                    cur_state = dest_state;
                    // 执行 pop 操作
                    for (int j = 0; j < max_pop_len && pop_list[j] != -1; ++j) {
                        --stack_top;
                    }
                    // 执行 push 操作
                    for (int j = 0; j < max_push_len && push_list[j] != -1; ++j) {
                        // stack[index(max_stack_depth, tid, stack_top)] = push_list[j];
                        stack[stack_top] = push_list[j];
                        stack_top++;
                    }

                    match = true;
                    break;
                }
            }

            if (!match) {
                cur_state = -1;  // 无法匹配，设置为无效状态
                break;
            }
        }

        // 存储最终的状态和栈指针
        output[batch_idx * num_seqs + seq_idx] = cur_state;
    }
}

void batched_check_dpda(
    torch::Tensor input_sequences,
    torch::Tensor sequence_len,
    torch::Tensor shift_table,
    torch::Tensor edge_num_table,
    torch::Tensor push_table,
    torch::Tensor pop_table,
    torch::Tensor dest_table,
    torch::Tensor current_stack,            // [batch_size, max_stack_depth]
    torch::Tensor current_stack_top,        // [batch_size]
    torch::Tensor start_state,              // [batch_size * num_seqs]
    torch::Tensor output,
    int max_stack_depth
) {
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    const int num_batch = current_stack_top.size(0);
    const int num_seqs = input_sequences.size(0);
    const int max_seq_len = input_sequences.size(1);
    const int num_node = edge_num_table.size(0);
    const int num_symbol = edge_num_table.size(1);
    const int max_edge_num_for_same_t = shift_table.size(2);
    const int max_pop_len = pop_table.size(1);
    const int max_push_len = push_table.size(1);

    // const int block_dim = (num_seqs + 255) / 256;
    // const int block_size = 256;
    // const int batch_size = current_stack.size(0);

    const int seq_per_thread = 32;
    const int block_size = seq_per_thread * 512;
    // printf("%d %d\n", num_seqs, sequence_len.size(0));
    dim3 grid(current_stack.size(0), (num_seqs + block_size - 1) / block_size, 1);

    batched_check_kernel<<<grid, block_size / seq_per_thread, 0, stream>>>(
        input_sequences.data_ptr<int>(),
        sequence_len.data_ptr<int>(),
        shift_table.data_ptr<int>(),
        edge_num_table.data_ptr<int>(),
        push_table.data_ptr<int>(),
        pop_table.data_ptr<int>(),
        dest_table.data_ptr<int>(),
        current_stack.data_ptr<int>(),
        start_state.data_ptr<int>(),
        current_stack_top.data_ptr<int>(),
        output.data_ptr<int>(),
        num_batch,
        num_seqs,
        num_symbol,
        num_node,
        max_seq_len,
        max_edge_num_for_same_t,
        max_stack_depth,
        max_pop_len,
        max_push_len,
        seq_per_thread,
        block_size
    );
}

PYBIND11_MODULE(lightllm_constraint_decode_kernel, m) {
  m.def("check_dpda", &check_dpda, "check_dpda");
  m.def("batched_check_dpda", &batched_check_dpda, "batched_check_dpda");
}
